#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

const int N = 1<<20;
const int THREADS = 256;

__global__ void vecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    size_t bytes = N * sizeof(float);
    float *hA, *hB, *hC, *hC_ref;
    hipHostMalloc(&hA, bytes, hipHostMallocDefault);
    hipHostMalloc(&hB, bytes, hipHostMallocDefault);
    hipHostMalloc(&hC, bytes, hipHostMallocDefault);
    hipHostMalloc(&hC_ref, bytes, hipHostMallocDefault);

    for (int i = 0; i < N; ++i) {
        hA[i] = static_cast<float>(i);
        hB[i] = static_cast<float>(2*i);
        hC_ref[i] = hA[i] + hB[i];
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);

    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);

    int blocks = (N + THREADS - 1) / THREADS;

    vecAdd<<<blocks, THREADS>>>(dA, dB, dC, N);
    hipDeviceSynchronize();

    const int RUNS = 10;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int r = 0; r < RUNS; ++r) {
        vecAdd<<<blocks, THREADS>>>(dA, dB, dC, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms /= RUNS;

    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);

    bool ok = true;
    const float tol = 1e-5f;
    for (int i = 0; i < N; ++i) {
        if (fabs(hC[i] - hC_ref[i]) > tol) {
            ok = false;
            break;
        }
    }

    printf("%s\n", ok ? "OK" : "FAIL");
    printf("TIME_MS %.3f\n", ms);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipHostFree(hC_ref);
    return 0;
}